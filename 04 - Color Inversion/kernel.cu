#include "hip/hip_runtime.h"
#include "solve.h"
#include <hip/hip_runtime.h>

__global__ void invert_kernel(unsigned char* image, int width, int height) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < width * height * 4) {
        image[idx] = 255 * ((idx + 1) % 4 != 0) - (image[idx] * (((idx + 1) % 4 != 0)*2-1));
    }   

}
// image_input, image_output are device pointers (i.e. pointers to memory on the GPU)
void solve(unsigned char* image, int width, int height) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (width * height + threadsPerBlock - 1) / threadsPerBlock;

    invert_kernel<<<blocksPerGrid, threadsPerBlock>>>(image, width, height);
    hipDeviceSynchronize();
}