#include "hip/hip_runtime.h"
#include "solve.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void matrix_transpose_kernel(const float* input, float* output, int rows, int cols) {
    int colIdx = blockDim.x * blockIdx.x + threadIdx.x;
    int rowIdx = blockDim.y * blockIdx.y + threadIdx.y;
    if ((colIdx < cols) && (rowIdx < rows)) {
        output[rows*colIdx + rowIdx] = input[cols*rowIdx+colIdx];
    }
}

// input, output are device pointers (i.e. pointers to memory on the GPU)
void solve(const float* input, float* output, int rows, int cols) {
    int BLOCK_SIZE = 32;
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocksPerGrid((cols + BLOCK_SIZE - 1) / BLOCK_SIZE,
                       (rows + BLOCK_SIZE - 1) / BLOCK_SIZE);

    matrix_transpose_kernel<<<blocksPerGrid, threadsPerBlock>>>(input, output, rows, cols);
    hipDeviceSynchronize();
}