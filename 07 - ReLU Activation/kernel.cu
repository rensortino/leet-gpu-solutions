#include "hip/hip_runtime.h"
#include "solve.h"
#include <hip/hip_runtime.h>

__global__ void relu_kernel(const float* input, float* output, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        // When idx is negative, the bool condition is 0
        output[idx] = input[idx] * !(input[idx] < 0);
    }
}

// input, output are device pointers (i.e. pointers to memory on the GPU)
void solve(const float* input, float* output, int N) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    relu_kernel<<<blocksPerGrid, threadsPerBlock>>>(input, output, N);
    hipDeviceSynchronize();
}
