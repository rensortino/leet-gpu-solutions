#include "hip/hip_runtime.h"
#include "solve.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void matrix_multiplication_kernel(const float* A, const float* B, float* C, int M, int N, int K) {
    int colIdx = blockDim.x * blockIdx.x + threadIdx.x;
    int rowIdx = blockDim.y * blockIdx.y + threadIdx.y;
    int rowMajIdx = rowIdx * K + colIdx;

    if ((colIdx < K) && (rowIdx < M)) {
        C[rowMajIdx] = 0.0;
        for (int i=0; i < N; i++) {
            C[rowMajIdx] += A[rowIdx * N + i] * B[colIdx + N * i];
        }
    }
}

// A, B, C are device pointers (i.e. pointers to memory on the GPU)
void solve(const float* A, const float* B, float* C, int M, int N, int K) {
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((K + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (M + threadsPerBlock.y - 1) / threadsPerBlock.y);
    
    matrix_multiplication_kernel<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, M, N, K);
    hipDeviceSynchronize();
}
